// Copyright 2023 Pierre Talbot


#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_world() {
  printf("Hello World!\n");
}

int main(int argc, char** argv) {
  hello_world<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}
